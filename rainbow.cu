#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <iostream>

const int blocksize = 16;

typedef struct {
  double3 coord;
  double r;
} sphere_t;

typedef struct {
  double3 coord;
  double3 dir;
  double wavelength;
} light_t;

typedef struct {
  light_t l;
  bool intersects;
} intersection_t;

/* Calculates the dot product of two 3D vectors */
__device__ double dot(double3 lhs, double3 rhs) {
  return lhs.x * rhs.x + lhs.y * rhs.y + lhs.z * rhs.z;
}

/* Multiply a 3D vector by a scalar */
__device__ double3 operator*(double3 vec, double scalar) {
  return {vec.x * scalar, vec.y * scalar, vec.z * scalar};
}

__device__ double3 operator*(double scalar, double3 vec) {
  return {vec.x * scalar, vec.y * scalar, vec.z * scalar};
}

/* Subtract two 3D vectors */
__device__ double3 operator-(double3 lhs, double3 rhs) {
  return {lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z};
}

/* Add two 3D vectors */
__device__ double3 operator+(double3 lhs, double3 rhs) {
  return {lhs.x + rhs.x, lhs.y + rhs.y, lhs.z + rhs.z};
}

/* Normalize a 3D vector */
__device__ double3 normalize(double3 v) {
  double magnitude = sqrt(pow(v.x, 2) + pow(v.y, 2) + pow(v.z, 2));
  return {v.x / magnitude, v.y / magnitude, v.z / magnitude};
}

/* Clamps a float between two values */
__device__ double clamp(double val, double lower, double upper) {
  return max(lower, min(val, upper));
}

/* Converts the wavelength in nm to the refractive index of the material, in
 * this case water-air */
__device__ double wavelengthToRefraction(double wavelength) {
  return 1.31477 + 0.0108148 / (log10(0.00690246 * wavelength));
}

/* Checks if the given point is in the sphere */
__device__ bool inSphere(sphere_t sphere, double3 coord) {
  double epsilon = 0.0001;

  return (pow((coord.x - sphere.coord.x), 2) +
          pow((coord.y - sphere.coord.y), 2) +
          pow((coord.z - sphere.coord.z), 2)) <= pow(sphere.r, 2) + epsilon;
}

/* Based on https://registry.khronos.org/OpenGL-Refpages/gl4/html/refract.xhtml
Given a normal vector, an incident vector, and a
wavelength, calculates the refracted vector */
__device__ double3 refract(double3 N, double3 I, double wavelength,
                           bool inWater) {
  double eta = wavelengthToRefraction(wavelength);
  if (!inWater) {
    eta = 1.0 / eta;
  }
  double k = 1.0 - eta * eta * (1.0 - dot(N, I) * dot(N, I));

  if (k < 0) {
    return {0, 0, 0};
  }

  return eta * I - (eta * dot(N, I) + sqrt(k)) * N;
}

/* Based on https://registry.khronos.org/OpenGL-Refpages/gl4/html/reflect.xhtml
Given an incident vector and a normal vector, calculates the reflected vector,
the normal vector must actually be normalzied for optimal results */
__device__ double3 reflect(double3 I, double3 N) {
  return I - 2 * dot(N, I) * N;
}

/* Calculates the intersections between a sphere and a radius, if there is
 * any*/
__device__ intersection_t vectorSphereIntersection(sphere_t s, light_t l) {
  /* Given the sphere's center coordinates and radius, and the radius's
   coordinates and direction, we calculate the intersection point:
   (x - s.x)^2 + (y - s.y)^2 + (z - s.z)^2 = s.r^2
   Where `x` is l.coord.x + t * l.dir.x, `y` is l.coord.y + t * l.dir.y,
   `z` is l.coord.z + t * l.dir.z (parametric equation, t is the parameter)
   We then solve for t, and use
   the discriminant to determine if there is an intersection or not.

   The fully expanded eqaution is:
   (l.dir.x^2 + l.dir.y^2 + l.dir.z^2) * t^2 +
   2 * (l.dir.x * (l.coord.x - s.coord.x) + l.dir.y * (l.coord.y - s.coord.y) +
   l.dir.z * (l.coord.z - s.coord.z)) * t +
   (l.coord.x - c.x)^2 + (l.coord.y - c.y)^2 + (l.coord.z - c.z)^2 - r^2 = 0 */

  // a = l.dir.x^2 + l.dir.y^2 + l.dir.z^2
  double a = pow(l.dir.x, 2) + pow(l.dir.y, 2) + pow(l.dir.z, 2);

  /* b = 2 * (l.dir.x * (l.coord.x - s.coord.x) +
  l.dir.y * (l.coord.y - s.coord.y) + l.dir.z * (l.coord.z - s.coord.z)) */
  double b = 2 * (l.dir.x * (l.coord.x - s.coord.x) +
                  l.dir.y * (l.coord.y - s.coord.y) +
                  l.dir.z * (l.coord.z - s.coord.z));

  // c = (l.coord.x - c.x)^2 + (l.coord.y - c.y)^2 + (l.coord.z - c.z)^2 - r^2
  double c = pow((l.coord.x - s.coord.x), 2) + pow((l.coord.y - s.coord.y), 2) +
             pow((l.coord.z - s.coord.z), 2) - pow(s.r, 2);

  // discriminant = b^2 - 4 * a * c
  double d = pow(b, 2) - 4 * a * c;

  // If the discriminant is negative, there is no solution
  intersection_t i;
  if (d < 0) {
    i.intersects = false;
    return i;
  }

  double t1 = (-1 * b + sqrt(d)) / (2 * a);
  double t2 = (-1 * b - sqrt(d)) / (2 * a);

  double epsilon = 0.0001;
  double t = 0;

  // If t1 is positive, is smaller than t2 or t2 is negative, we use t1
  // If t2 is positive, is smaller than t1 or t1 is negative, we use t2
  // If both are negative, there is no intersection
  if (0 < t1 && (t1 < t2 || abs(t2) <= epsilon)) {
    i.intersects = true;
    t = t1;
  } else if (0 < t2 && (t2 < t1 || abs(t1) <= epsilon)) {
    i.intersects = true;
    t = t2;
  } else {
    i.intersects = false;
  }

  if (i.intersects) {
    i.l = {l.coord.x + t * l.dir.x,
           l.coord.y + t * l.dir.y,
           l.coord.z + t * l.dir.z,
           l.dir.x,
           l.dir.y,
           l.dir.z};
  }

  return i;
}

/* Calculates the normal vector for a sphere and intersection point */
__device__ double3 calculateNormalVector(sphere_t s, double3 i) {
  /* Given a sphere and a point on the sphere's surface, calculate the
   * vector from the sphere's center to the intersection point */
  double3 vector = {i.x - s.coord.x, i.y - s.coord.y, i.z - s.coord.z};

  /* Normalize the given vector */
  return normalize(vector);
}

/* Calculates the angle between two 3D vectors */
__device__ double angleBetweenVectors(double3 lhs, double3 rhs) {
  /* Calculate the dot product of the vectors */
  double dotProduct = dot(lhs, rhs);

  /* Calculate the magnitudes of the vectors */
  double magnL = sqrt(pow(lhs.x, 2) + pow(lhs.y, 2) + pow(lhs.z, 2));
  double magnR = sqrt(pow(rhs.x, 2) + pow(rhs.y, 2) + pow(rhs.z, 2));

  /* Calculate the angle's cosine between the vectors */
  double cosA = dotProduct / (magnL * magnR);

  /* Return the radians in degrees of the angle between the vectors */
  return acos(cosA);
}

__device__ int getGlobalIdx_2D_2D() {
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int threadId = blockId * (blockDim.x * blockDim.y) +
                 (threadIdx.y * blockDim.x) + threadIdx.x;
  return threadId;
}

__global__ void rainbowAirWater(double *wavelength, light_t *returnVal) {
  unsigned int idx = getGlobalIdx_2D_2D();

  sphere_t sphere = {{2, -2, 1}, 3};
  light_t light = {{3, 2, -3}, {0, -1, 1}, wavelength[idx]};

  intersection_t intersection = vectorSphereIntersection(sphere, light);

  bool refraction = true;
  bool inWater = false;

  returnVal[idx] = light;

  for (int i = 0; i < 4 && inSphere(sphere, intersection.l.coord) &&
                  intersection.intersects;
       ++i) {
    double3 normalVector = calculateNormalVector(sphere, intersection.l.coord);
    float angle = clamp(
        angleBetweenVectors(light.dir, intersection.l.coord + normalVector), 0,
        M_PI / 2);

    double3 newVector;
    if (refraction) {
      if (inWater) {
        normalVector = -1 * normalVector;
      }

      newVector =
          refract(normalVector, intersection.l.dir, light.wavelength, inWater);
      inWater = true;
      refraction = false;
    } else {
      newVector = reflect(intersection.l.dir, -1 * normalVector);
      refraction = true;
    }
    light = {intersection.l.coord, newVector, light.wavelength};
    intersection = vectorSphereIntersection(sphere, light);
  }

  returnVal[idx] = light;
}
}

int main() {
  const int WAVELENGTHS = 680 - 380;
  double wavelength[WAVELENGTHS];
  for (int i = 0; i < WAVELENGTHS; ++i) {
    wavelength[i] = 380 + i;
  }

  double *gpu_wavelength;

  dim3 block_size(4, 4);
  dim3 grid_size(5, 5);

  light_t *cpu_results, *gpu_results;

  float *hostVal = 0;
  float *val;

  hipError_t hipError_t = hipMalloc((void **)&val, 3 * sizeof(float));
  if (hipError_t != hipSuccess) {
    std::cout << "Error while allocating memory on GPU: "
              << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }

  hipError_t =
      hipHostAlloc((void **)&cpu_results, WAVELENGTHS * sizeof(light_t),
                    hipHostMallocDefault);
  if (hipError_t != hipSuccess) {
    std::cout << "Error while allocating pinned memory: "
              << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }

  hipError_t = hipHostAlloc((void **)&gpu_wavelength,
                            WAVELENGTHS * sizeof(double), hipHostMallocDefault);
  if (hipError_t != hipSuccess) {
    std::cout << "Error while allocating pinned memory: "
              << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }
  hipMemcpy(gpu_wavelength, wavelength, WAVELENGTHS * sizeof(double),
             hipMemcpyHostToDevice);

  auto tS = std::chrono::high_resolution_clock::now();

  rainbowAirWater<<<block_size, grid_size>>>(gpu_wavelength, gpu_results);

  hipMemcpy(cpu_results, gpu_results, WAVELENGTHS * sizeof(light_t),
             hipMemcpyDeviceToHost);

  auto diff = std::chrono::high_resolution_clock::now() - tS;
  std::cout << (ulong)std::chrono::duration_cast<std::chrono::microseconds>(
                   diff)
                   .count()
            << std::endl;

  // for (int i = 0; i < WAVELENGTHS; ++i) {
  //   std::cout << cpu_results[i].wavelength << "nm (" <<
  //   cpu_results[i].coord.x
  //             << ", " << cpu_results[i].coord.y << ", "
  //             << cpu_results[i].coord.z << ") "
  //             << " -> "
  //             << "(" << cpu_results[i].dir.x << ", " << cpu_results[i].dir.y
  //             << ", " << cpu_results[i].dir.z << ")" << std::endl;
  // }
  hipHostFree(cpu_rgb);
  hipHostFree(cpu_results);
  hipHostFree(gpu_wavelength);
  hipFree(gpu_results);

  return EXIT_SUCCESS;
}