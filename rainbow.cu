#include "hip/hip_runtime.h"
#include <cstdlib>
#include <ostream>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <chrono>
#include <cmath>
#include <iostream>

#define CHANNEL_NUM 3
typedef struct {
  double3 coord;
  double r;
} sphere_t;

typedef struct {
  double3 coord;
  double3 dir;
  double wavelength;
} light_t;

typedef struct {
  light_t l;
  bool intersects;
} intersection_t;

/* Calculates the dot product of two 3D vectors */
__device__ double dot(double3 lhs, double3 rhs) {
  return lhs.x * rhs.x + lhs.y * rhs.y + lhs.z * rhs.z;
}

/* Multiply a 3D vector by a scalar */
__device__ double3 operator*(double3 vec, double scalar) {
  return {vec.x * scalar, vec.y * scalar, vec.z * scalar};
}

__device__ double3 operator*(double scalar, double3 vec) {
  return {vec.x * scalar, vec.y * scalar, vec.z * scalar};
}

/* Subtract two 3D vectors */
__device__ double3 operator-(double3 lhs, double3 rhs) {
  return {lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z};
}

/* Add two 3D vectors */
__device__ double3 operator+(double3 lhs, double3 rhs) {
  return {lhs.x + rhs.x, lhs.y + rhs.y, lhs.z + rhs.z};
}

/* Normalize a 3D vector */
__device__ double3 normalize(double3 v) {
  double magnitude = sqrt(pow(v.x, 2) + pow(v.y, 2) + pow(v.z, 2));
  return {v.x / magnitude, v.y / magnitude, v.z / magnitude};
}

/* Clamps a float between two values */
__device__ double clamp(double val, double lower, double upper) {
  return max(lower, min(val, upper));
}

/* Converts the wavelength in nm to the refractive index of the material, in
 * this case water-air */
__device__ double wavelengthToRefraction(double wavelength) {
  return 1.31477 + 0.0108148 / (log10(0.00690246 * wavelength));
}

/* Checks if the given point is in the sphere */
__device__ bool inSphere(sphere_t sphere, double3 coord) {
  double epsilon = 0.0001;

  return (pow((coord.x - sphere.coord.x), 2) +
          pow((coord.y - sphere.coord.y), 2) +
          pow((coord.z - sphere.coord.z), 2)) <= pow(sphere.r, 2) + epsilon;
}

/* Based on https://registry.khronos.org/OpenGL-Refpages/gl4/html/refract.xhtml
Given a normal vector, an incident vector, and a
wavelength, calculates the refracted vector */
__device__ double3 refract(double3 N, double3 I, double wavelength,
                           bool inWater) {
  double eta = wavelengthToRefraction(wavelength);
  if (!inWater) {
    eta = 1.0 / eta;
  }
  double k = 1.0 - eta * eta * (1.0 - dot(N, I) * dot(N, I));

  if (k < 0) {
    return {0, 0, 0};
  }

  return eta * I - (eta * dot(N, I) + sqrt(k)) * N;
}

/* Based on https://registry.khronos.org/OpenGL-Refpages/gl4/html/reflect.xhtml
Given an incident vector and a normal vector, calculates the reflected vector,
the normal vector must actually be normalzied for optimal results */
__device__ double3 reflect(double3 I, double3 N) {
  return I - 2 * dot(N, I) * N;
}

/* Calculates the intersections between a sphere and a radius, if there is
 * any*/
__device__ intersection_t vectorSphereIntersection(sphere_t s, light_t l) {
  /* Given the sphere's center coordinates and radius, and the radius's
   coordinates and direction, we calculate the intersection point:
   (x - s.x)^2 + (y - s.y)^2 + (z - s.z)^2 = s.r^2
   Where `x` is l.coord.x + t * l.dir.x, `y` is l.coord.y + t * l.dir.y,
   `z` is l.coord.z + t * l.dir.z (parametric equation, t is the parameter)
   We then solve for t, and use
   the discriminant to determine if there is an intersection or not.

   The fully expanded eqaution is:
   (l.dir.x^2 + l.dir.y^2 + l.dir.z^2) * t^2 +
   2 * (l.dir.x * (l.coord.x - s.coord.x) + l.dir.y * (l.coord.y - s.coord.y) +
   l.dir.z * (l.coord.z - s.coord.z)) * t +
   (l.coord.x - c.x)^2 + (l.coord.y - c.y)^2 + (l.coord.z - c.z)^2 - r^2 = 0 */

  // a = l.dir.x^2 + l.dir.y^2 + l.dir.z^2
  double a = pow(l.dir.x, 2) + pow(l.dir.y, 2) + pow(l.dir.z, 2);

  /* b = 2 * (l.dir.x * (l.coord.x - s.coord.x) +
  l.dir.y * (l.coord.y - s.coord.y) + l.dir.z * (l.coord.z - s.coord.z)) */
  double b = 2 * (l.dir.x * (l.coord.x - s.coord.x) +
                  l.dir.y * (l.coord.y - s.coord.y) +
                  l.dir.z * (l.coord.z - s.coord.z));

  // c = (l.coord.x - c.x)^2 + (l.coord.y - c.y)^2 + (l.coord.z - c.z)^2 - r^2
  double c = pow((l.coord.x - s.coord.x), 2) + pow((l.coord.y - s.coord.y), 2) +
             pow((l.coord.z - s.coord.z), 2) - pow(s.r, 2);

  // discriminant = b^2 - 4 * a * c
  double d = pow(b, 2) - 4 * a * c;

  // If the discriminant is negative, there is no solution
  intersection_t i;
  if (d < 0) {
    i.intersects = false;
    return i;
  }

  double t1 = (-1 * b + sqrt(d)) / (2 * a);
  double t2 = (-1 * b - sqrt(d)) / (2 * a);

  double epsilon = 0.0001;
  double t = 0;

  // If t1 is positive, is smaller than t2 or t2 is negative, we use t1
  // If t2 is positive, is smaller than t1 or t1 is negative, we use t2
  // If both are negative, there is no intersection
  if (0 < t1 && (t1 < t2 || abs(t2) <= epsilon)) {
    i.intersects = true;
    t = t1;
  } else if (0 < t2 && (t2 < t1 || abs(t1) <= epsilon)) {
    i.intersects = true;
    t = t2;
  } else {
    i.intersects = false;
  }

  if (i.intersects) {
    i.l = {l.coord.x + t * l.dir.x,
           l.coord.y + t * l.dir.y,
           l.coord.z + t * l.dir.z,
           l.dir.x,
           l.dir.y,
           l.dir.z};
  }

  return i;
}

/* Calculates the normal vector for a sphere and intersection point */
__device__ double3 calculateNormalVector(sphere_t s, double3 i) {
  /* Given a sphere and a point on the sphere's surface, calculate the
   * vector from the sphere's center to the intersection point */
  double3 vector = {i.x - s.coord.x, i.y - s.coord.y, i.z - s.coord.z};

  /* Normalize the given vector */
  return normalize(vector);
}

/* Calculates the angle between two 3D vectors */
__device__ double angleBetweenVectors(double3 lhs, double3 rhs) {
  /* Calculate the dot product of the vectors */
  double dotProduct = dot(lhs, rhs);

  /* Calculate the magnitudes of the vectors */
  double magnL = sqrt(pow(lhs.x, 2) + pow(lhs.y, 2) + pow(lhs.z, 2));
  double magnR = sqrt(pow(rhs.x, 2) + pow(rhs.y, 2) + pow(rhs.z, 2));

  /* Calculate the angle's cosine between the vectors */
  double cosA = dotProduct / (magnL * magnR);

  /* Return the radians in degrees of the angle between the vectors */
  return acos(cosA);
}

__device__ int getGlobalIdx_2D_2D() {
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int threadId = blockId * (blockDim.x * blockDim.y) +
                 (threadIdx.y * blockDim.x) + threadIdx.x;
  return threadId;
}

__global__ void rainbowAirWater(double *wavelength, light_t *returnVal) {
  unsigned int idx = getGlobalIdx_2D_2D();

  sphere_t sphere = {{2, -2, 1}, 3};
  light_t light = {{3, 2, -3}, {0, -1, 1}, wavelength[idx]};

  intersection_t intersection = vectorSphereIntersection(sphere, light);

  bool refraction = true;
  bool inWater = false;

  for (int i = 0; i < 4 && inSphere(sphere, intersection.l.coord) &&
                  intersection.intersects;
       ++i) {
    double3 normalVector = calculateNormalVector(sphere, intersection.l.coord);
    float angle = clamp(
        angleBetweenVectors(light.dir, intersection.l.coord + normalVector), 0,
        M_PI / 2);

    double3 newVector;
    if (refraction) {
      if (inWater) {
        normalVector = -1 * normalVector;
      }

      newVector =
          refract(normalVector, intersection.l.dir, light.wavelength, inWater);
      inWater = true;
      refraction = false;
    } else {
      newVector = reflect(intersection.l.dir, -1 * normalVector);
      refraction = true;
    }
    light = {intersection.l.coord, newVector, light.wavelength};
    intersection = vectorSphereIntersection(sphere, light);
  }

  returnVal[idx] = light;
}

__global__ void wavelengthToRGB(double *wavelength, int3 *rgb) {
  unsigned int idx = getGlobalIdx_2D_2D();

  double gamma = 0.80;
  double intensityMax = 255;

  double factor;
  float3 curr_rgb;

  if ((wavelength[idx] >= 380) && (wavelength[idx] < 440)) {
    curr_rgb.x = -(wavelength[idx] - 440) / (440 - 380);
    curr_rgb.y = 0.0;
    curr_rgb.z = 1.0;
  } else if ((wavelength[idx] >= 440) && (wavelength[idx] < 490)) {
    curr_rgb.x = 0.0;
    curr_rgb.y = (wavelength[idx] - 440) / (490 - 440);
    curr_rgb.z = 1.0;
  } else if ((wavelength[idx] >= 490) && (wavelength[idx] < 510)) {
    curr_rgb.x = 0.0;
    curr_rgb.y = 1.0;
    curr_rgb.z = -(wavelength[idx] - 510) / (510 - 490);
  } else if ((wavelength[idx] >= 510) && (wavelength[idx] < 580)) {
    curr_rgb.x = (wavelength[idx] - 510) / (580 - 510);
    curr_rgb.y = 1.0;
    curr_rgb.z = 0.0;
  } else if ((wavelength[idx] >= 580) && (wavelength[idx] < 645)) {
    curr_rgb.x = 1.0;
    curr_rgb.y = -(wavelength[idx] - 645) / (645 - 580);
    curr_rgb.z = 0.0;
  } else if ((wavelength[idx] >= 645) && (wavelength[idx] < 781)) {
    curr_rgb.x = 1.0;
    curr_rgb.y = 0.0;
    curr_rgb.z = 0.0;
  } else {
    curr_rgb.x = 0.0;
    curr_rgb.y = 0.0;
    curr_rgb.z = 0.0;
  }

  if ((wavelength[idx] >= 380) && (wavelength[idx] < 420)) {
    factor = 0.3 + 0.7 * (wavelength[idx] - 380) / (420 - 380);
  } else if ((wavelength[idx] >= 420) && (wavelength[idx] < 701)) {
    factor = 1.0;
  } else if ((wavelength[idx] >= 701) && (wavelength[idx] < 781)) {
    factor = 0.3 + 0.7 * (780 - wavelength[idx]) / (780 - 700);
  } else {
    factor = 0.0;
  }

  rgb[idx] = {curr_rgb.x == 0
                  ? 0
                  : (int)round(intensityMax * pow(curr_rgb.x * factor, gamma)),
              curr_rgb.y == 0
                  ? 0
                  : (int)round(intensityMax * pow(curr_rgb.y * factor, gamma)),
              curr_rgb.z == 0
                  ? 0
                  : (int)round(intensityMax * pow(curr_rgb.z * factor, gamma))};
}

int main() {
  // Wavelengths from 380nm to 680nm on the CPU
  const int WAVELENGTHS = 680 - 380;
  double wavelength[WAVELENGTHS];
  for (int i = 0; i < WAVELENGTHS; ++i) {
    wavelength[i] = 380 + i;
  }

  // Variables for use on the GPU & CPU
  double *gpu_wavelength;
  light_t *cpu_results, *gpu_results;
  int3 *cpu_rgb, *gpu_rgb;

  // Grid & Block size for the kernel, 3 * 4 * 5 * 5 = 300 threads, the 300
  // wavelengths
  dim3 block_size(3, 4);
  dim3 grid_size(5, 5);

  // Allocate memory on the GPU & CPU
  hipError_t hipError_t =
      hipMalloc((void **)&gpu_results, WAVELENGTHS * sizeof(light_t));
  if (hipError_t != hipSuccess) {
    std::cout << "Error while allocating memory on GPU: "
              << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }

  hipError_t =
      hipHostAlloc((void **)&cpu_results, WAVELENGTHS * sizeof(light_t),
                    hipHostMallocDefault);
  if (hipError_t != hipSuccess) {
    std::cout << "Error while allocating pinned memory: "
              << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }

  hipError_t =
      hipMalloc((void **)&gpu_wavelength, WAVELENGTHS * sizeof(double));
  if (hipError_t != hipSuccess) {
    std::cout << "Error while allocating pinned memory: "
              << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }
  hipMemcpy(gpu_wavelength, wavelength, WAVELENGTHS * sizeof(double),
             hipMemcpyHostToDevice);

  hipError_t = hipMalloc((void **)&gpu_rgb, WAVELENGTHS * sizeof(int3));
  if (hipError_t != hipSuccess) {
    std::cout << "Error while allocating pinned memory: "
              << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }

  hipError_t = hipHostAlloc((void **)&cpu_rgb, WAVELENGTHS * sizeof(int3),
                            hipHostMallocDefault);
  if (hipError_t != hipSuccess) {
    std::cout << "Error while allocating pinned memory: "
              << hipGetErrorString(hipError_t) << std::endl;
    exit(1);
  }

  // Clock for timing
  // auto tS = std::chrono::high_resolution_clock::now();

  // Run the kernel for the rainbow vector calculation
  rainbowAirWater<<<block_size, grid_size>>>(gpu_wavelength, gpu_results);

  // Copy back the result to the CPU
  hipMemcpy(cpu_results, gpu_results, WAVELENGTHS * sizeof(light_t),
             hipMemcpyDeviceToHost);

  // Print the timing
  // auto diff = std::chrono::high_resolution_clock::now() - tS;
  // std::cout << (ulong)std::chrono::duration_cast<std::chrono::microseconds>(
  //                  diff)
  //                  .count()
  //           << std::endl;

  // Print the refraction, reflection results
  // for (int i = 0; i < WAVELENGTHS; ++i) {
  //   std::cout << cpu_results[i].wavelength << "nm (" <<
  //   cpu_results[i].coord.x
  //             << ", " << cpu_results[i].coord.y << ", "
  //             << cpu_results[i].coord.z << ") "
  //             << " -> "
  //             << "(" << cpu_results[i].dir.x << ", " << cpu_results[i].dir.y
  //             << ", " << cpu_results[i].dir.z << ")" << std::endl;
  // }

  // Run the kernel for the wavelength -> RGB conversion
  wavelengthToRGB<<<block_size, grid_size>>>(gpu_wavelength, gpu_rgb);

  // Copy back the result to the CPU
  hipMemcpy(cpu_rgb, gpu_rgb, WAVELENGTHS * sizeof(int3),
             hipMemcpyDeviceToHost);

  /* Calculate the image, since we are using the z = -3.0 plane, our x values
   * will range from -1.94345 (Ultraviolet light) to -1.9854 (Red light)
   * Since this change from -1.94345 to -1.9854 is 300 (the number of
   * wavelengths), we can calculate the resolution for -1.90 to -2.00, which is
   * 715 300 is used for the height, since we take 3 pixel at just before 2.0,
   * at 2.0 and just after 2.0 for the y value
   */
  const int WIDTH = 715;
  const int HEIGHT = 300;

  unsigned char *pixels = new unsigned char[WIDTH * HEIGHT * CHANNEL_NUM];
  memset(pixels, 255, WIDTH * HEIGHT * CHANNEL_NUM * sizeof(unsigned char));

  double zPlane = -3.0;
  // Used for indexing the pixel array
  int idx = 0;
  for (int j = 0; j < HEIGHT; ++j) {
    for (int i = 0; i < WIDTH; ++i) {
      // Current x and y values based on the resolution we specified
      double currentX = -1.90 + (i * -0.000140161);
      double currentY = 1.9 + (int)(j / 100) * 0.1;
      double epsilon = 0.001;
      bool inRange = false;
      /* Iterate over the vector results and find the vector that intersects
       * the plane at the current x and y values (if any) */
      for (int k = 0; k < WAVELENGTHS; ++k) {
        /* t is used for the parametric equation of the line
         * we need to calculate the t such that
         * cpu_results[k].coord.z + t * cpu_results[k].dir.z = -3.0 */
        double t = (zPlane - cpu_results[k].coord.z) / cpu_results[k].dir.z;
        /* If for the given t, the x and y values are within epsilon of the
         * current calculated x and y values, we take the wavelength of the
         * vector and convert it to RGB */
        if (abs(cpu_results[k].coord.x + t * cpu_results[k].dir.x - currentX) <=
                epsilon &&
            abs(cpu_results[k].coord.y + t * cpu_results[k].dir.y - currentY) <=
                epsilon) {
          pixels[idx++] = cpu_rgb[(int)cpu_results[k].wavelength - 380].x;
          pixels[idx++] = cpu_rgb[(int)cpu_results[k].wavelength - 380].y;
          pixels[idx++] = cpu_rgb[(int)cpu_results[k].wavelength - 380].z;
          inRange = true;
          break;
        }
      }
      if (!inRange) {
        // Else we use white
        pixels[idx++] = 255;
        pixels[idx++] = 255;
        pixels[idx++] = 255;
      }
    }
  }

  // Write the image to the file
  stbi_write_png("rainbow.png", WIDTH, HEIGHT, CHANNEL_NUM, pixels,
                 WIDTH * CHANNEL_NUM);

  // Free up memory
  hipHostFree(cpu_rgb);
  hipHostFree(cpu_results);
  hipFree(gpu_wavelength);
  hipFree(gpu_results);
  hipFree(gpu_rgb);

  return EXIT_SUCCESS;
}