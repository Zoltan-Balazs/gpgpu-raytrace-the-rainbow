#include "hip/hip_runtime.h"
#include <stdio.h>

const int N = 16;
const int blocksize = 16;

typedef struct {
  float3 coordinates;
  float radius;
} sphere_t;

typedef struct {
  float3 coordinates;
  float3 direction;
  double wavelength;
} light_t;

typedef struct {
  light_t light;
  bool intersects;
} intersection_t;

/* Converts the wavelength in nm to the refractive index of the material, in
 * this case water-air */
__device__ double wavelengthToRefraction(double wavelength) {
  return 1.31477 + 0.0108148 / (log10(0.00690246 * wavelength));
}

/* Checks if the given point is in the sphere */
__device__ bool inSphere(sphere_t sphere, float3 coordinate) {
  double epsilon = 0.0001;

  return abs((sphere.radius * sphere.radius) -
             (pow((coordinate.x - sphere.coordinates.x), 2) +
              pow((coordinate.y - sphere.coordinates.y), 2) +
              pow((coordinate.z - sphere.coordinates.z), 2))) <= epsilon;
}

/* Calculates the dot product of two 3D vectors */
__device__ float dot(float3 a, float3 b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

/* Multiply a 3D vector by a scalar */
__device__ float3 operator*(float3 vec, float scalar) {
  return {vec.x * scalar, vec.y * scalar, vec.z * scalar};
}
int main() {
  char a[N] = "Hello \0\0\0\0\0\0";
  int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

  float3 *world;
  (void)world;

  char *ad;
  int *bd;
  const int csize = N * sizeof(char);
  const int isize = N * sizeof(int);

  printf("%s", a);

  hipMalloc((void **)&ad, csize);
  hipMalloc((void **)&bd, isize);
  hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
  hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);
  hello<<<dimGrid, dimBlock>>>(ad, bd);
  hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
  hipFree(ad);
  hipFree(bd);

  printf("%s\n", a);
  return EXIT_SUCCESS;
}