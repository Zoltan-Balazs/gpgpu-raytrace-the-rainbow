#include "hip/hip_runtime.h"
#include <stdio.h>

const int N = 16;
const int blocksize = 16;

typedef struct {
  float3 coordinates;
  float radius;
} sphere_t;

typedef struct {
  float3 coordinates;
  float3 direction;
  double wavelength;
} light_t;

typedef struct {
  light_t light;
  bool intersects;
} intersection_t;

/* Calculates the dot product of two 3D vectors */
__device__ float dot(float3 lhs, float3 rhs) {
  return lhs.x * rhs.x + lhs.y * rhs.y + lhs.z * rhs.z;
}

/* Multiply a 3D vector by a scalar */
__device__ float3 operator*(float3 vec, float scalar) {
  return {vec.x * scalar, vec.y * scalar, vec.z * scalar};
}

__device__ float3 operator*(float scalar, float3 vec) {
  return {vec.x * scalar, vec.y * scalar, vec.z * scalar};
}

/* Subtract two 3D vectors */
__device__ float3 operator-(float3 lhs, float3 rhs) {
  return {lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z};
}

/* Normalize a 3D vector */
__device__ float3 normalize(float3 v) {
  float magnitude = sqrt(pow(v.x, 2) + pow(v.y, 2) + pow(v.z, 2));
  return {v.x / magnitude, v.y / magnitude, v.z / magnitude};
}

/* Converts the wavelength in nm to the refractive index of the material, in
 * this case water-air */
__device__ double wavelengthToRefraction(double wavelength) {
  return 1.31477 + 0.0108148 / (log10(0.00690246 * wavelength));
}

/* Checks if the given point is in the sphere */
__device__ bool inSphere(sphere_t sphere, float3 coordinate) {
  double epsilon = 0.0001;

  return abs((sphere.radius * sphere.radius) -
             (pow((coordinate.x - sphere.coordinates.x), 2) +
              pow((coordinate.y - sphere.coordinates.y), 2) +
              pow((coordinate.z - sphere.coordinates.z), 2))) <= epsilon;
}

/* Based on https://registry.khronos.org/OpenGL-Refpages/gl4/html/refract.xhtml
Given a normal vector, an incident vector, and a
wavelength, calculates the refracted vector */
__device__ float3 refract(float3 N, float3 I, double wavelength) {
  float eta = wavelengthToRefraction(wavelength);
  eta = 1.0 / eta;
  float k = 1.0 - eta * eta * (1.0 - dot(N, I) * dot(N, I));

  if (k < 0) {
    return {0, 0, 0};
  }

  return eta * I - N * (eta * dot(N, I) + sqrt(k));
}

/* Based on https://registry.khronos.org/OpenGL-Refpages/gl4/html/reflect.xhtml
Given an incident vector and a normal vector, calculates the reflected vector,
the normal vector must actually be normalzied for optimal results */
__device__ float3 reflect(float3 I, float3 N) {
  return I - 2 * dot(normalize(N), I) * normalize(N);
}
}

/* Calculates the normal vector for a sphere and intersection point */
__device__ float3 calculateNormalVector(sphere_t sphere,
                                        float3 intersectionPoint) {
  /* Given a sphere and a point on the sphere's surface, calculate the
   * vector from the sphere's center to the intersection point */
  float3 vector = {intersectionPoint.x - sphere.coordinates.x,
                   intersectionPoint.y - sphere.coordinates.y,
                   intersectionPoint.z - sphere.coordinates.z};

  /* Normalize the given vector */
  return normalize(vector);
}

/* Calculates the angle between two 3D vectors */
__device__ float angleBetweenVectors(float3 lhs, float3 rhs) {
  /* Calculate the dot product of the vectors */
  float dotProduct = dot(lhs, rhs);

  /* Calculate the magnitudes of the vectors */
  float magnL = sqrt(pow(lhs.x, 2) + pow(lhs.y, 2) + pow(lhs.z, 2));
  float magnR = sqrt(pow(rhs.x, 2) + pow(rhs.y, 2) + pow(rhs.z, 2));

  /* Calculate the angle's cosine between the vectors */
  float cosA = dotProduct / (magnL * magnR);

  /* Return the radians in degrees of the angle between the vectors */
  return acos(cosA);
}

/* Based on https://registry.khronos.org/OpenGL-Refpages/gl4/html/refract.xhtml
Given a normal vector, an incident vector, and a
wavelength, calculates the refracted vector */
__device__ float3 refract(float3 N, float3 I, double wavelength) {
  float eta = wavelengthToRefraction(wavelength);
  eta = 1.0 / eta;
  float k = 1.0 - eta * eta * (1.0 - dot(N, I) * dot(N, I));

  if (k < 0) {
    return {0, 0, 0};
  }

  return eta * I - N * (eta * dot(N, I) + sqrt(k));
}

/* Based on https://registry.khronos.org/OpenGL-Refpages/gl4/html/reflect.xhtml
Given an incident vector and a normal vector, calculates the reflected vector,
the normal vector must actually be normalzied for optimal results */
__device__ float3 reflect(float3 I, float3 N) { return I - 2 * dot(N, I) * N; }

  float3 *world;
  (void)world;

  char *ad;
  int *bd;
  const int csize = N * sizeof(char);
  const int isize = N * sizeof(int);

  printf("%s", a);

  hipMalloc((void **)&ad, csize);
  hipMalloc((void **)&bd, isize);
  hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
  hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);
  hello<<<dimGrid, dimBlock>>>(ad, bd);
  hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
  hipFree(ad);
  hipFree(bd);

  printf("%s\n", a);
  return EXIT_SUCCESS;
}